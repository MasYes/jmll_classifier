
#include <hip/hip_runtime.h>
extern "C"
__global__ void dropoutTrain(
    const float* arguments,
    float* dropoutMask,
    float* results,
    const float dropoutFraction,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    const float mask = dropoutFraction < dropoutMask[index];
    dropoutMask[index] = mask;
    results[index] = mask * arguments[index];
  }
}

extern "C"
__global__ void dropoutTest(
    const float* arguments,
    float* results,
    const float dropoutFraction,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = arguments[index] * (1.f - dropoutFraction);
  }
}
