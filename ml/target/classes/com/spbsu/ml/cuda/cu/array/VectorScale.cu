
#include <hip/hip_runtime.h>
extern "C"
__global__ void fSigmoid(
    const float* arguments,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = 1.f / (1.f + expf(-arguments[index]));
  }
}

extern "C"
__global__ void fDerSigmoid(
    const float* arguments,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    const float argument = arguments[index];
    results[index] = argument - argument * argument;
  }
}

extern "C"
__global__ void fExp(
    const float* arguments,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = expf(arguments[index]);
  }
}

extern "C"
__global__ void fTanh(
    const float* arguments,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = tanh(arguments[index]);
  }
}

extern "C"
__global__ void fNegation(
    const float* arguments,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = -arguments[index];
  }
}

extern "C"
__global__ void fHadamard(
    const float* argumentsA,
    const float* argumentsB,
    float* results,
    const long size
) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    results[index] = argumentsA[index] * argumentsB[index];
  }
}
